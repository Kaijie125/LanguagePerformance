
#include <hip/hip_runtime.h>

#define BLOCK_SIZE_X 256
#define BLOCK_SIZE_Y 1

__constant__ int dev_iter;
__constant__ double dev_x_min;
__constant__ double dev_y_min;
__constant__ double dev_x_step;
__constant__ double dev_y_step;
__constant__ int dev_W_x_H;
__constant__ int dev_width;
__constant__ int dev_row_size;

// colorea el pixel en funcion del numero de iteraciones
__device__ void setPixelColor(int iter, double re2, double im2, char *r, char *g, char *b){
    
    // escala de grises que es menos exigente
    int gray = (char)(255 * iter / dev_iter);
    *r = gray;
    *g = gray;
    *b = gray;

    /*
    double mu = iter + 1 - log(log(sqrt(re2 + im2))) / log(2.0);
    double log_zn = log(re2 + im2) / 2.0;
    double nu = log(log_zn / log(2.0)) / log(2.0);
    double iter_smooth = iter + 1 - nu;

    double t = iter_smooth / 256.0;
    double t2 = t * t;
    double t3 = t2 * t;
    double t4 = t3 * t;
    double t5 = t4 * t;

    *r = (int)(9 * (1 - t5) * t4 * 255);
    *g = (int)(15 * (1 - t5) * t3 * t2 * 255);
    *b = (int)(8.5 * (1 - t5) * t2 * t * 255);
    */
}

// calcula las itereaciones de mandelbrot -->  Z_{n+1} = Z_n^2 + C  
__device__ void pixelIter(int i, int j, char *r, char *g, char *b){
    
    int iteration = 0;
    
    // pixel a coordenadas complejas, C de la formula 
    //donde se suma medio "step" para estar en el centro del pixel
    double c_re = dev_x_min + dev_x_step/2 + j * dev_x_step;
    double c_im = dev_y_min + dev_y_step/2 + i * dev_y_step;
    //Z_0 = 0
    double zn_re = 0;
    double zn_im = 0;
    //temporal
    double tmp_re;
    // valor de Z_n^2 inicialmente es 0 + 0i
    double re2 = 0;
    double im2 = 0;

    while ((re2 + im2 < 4) && (iteration < dev_iter)){
        // Z_{n+1} = Z_n^2 + C => z_n+1_R = z_n_R^2 - z_n_I^2 + c_R
        //                        z_n+1_I = 2 * z_n_R * z_n_I + c_I
        tmp_re = re2 - im2 + c_re;
        zn_im = 2 * zn_re * zn_im + c_im;
        zn_re = tmp_re;

        //valores para la siguiente iteracion
        re2 = zn_re * zn_re;
        im2 = zn_im * zn_im;
        iteration++;
    }
    
    setPixelColor(iteration, re2, im2, r, g, b);
}


// funcion principal de mandelbrot
__global__ void mandelbrot(char *dev_img){
    // calcula el offset del pixel mediante el indice del bloque y el indice del thread
    int offset = gridDim.x * blockDim.x * threadIdx.y + blockIdx.x * blockDim.x + threadIdx.x;
    int i = offset / dev_width;
    int j = offset - i * dev_width;

    // comprueba que el pixel esta dentro de la imagen
    if (offset < dev_W_x_H){
        offset = dev_row_size * i + 3 * j; // offset in the image array
        
        pixelIter(i, j, &dev_img[offset + 2], &dev_img[offset + 1], &dev_img[offset]);

    }
}

void kernel_mandelbrot(char *hst_img, int img_size, int iter, double x_min, double y_min, double x_step, double y_step, int W_x_H, int width, int row_size){
    // puntero a la imagen en el device
    char *dev_img;

    // dimensiones de los bloques y de la grid
    dim3 block_size;
    dim3 grid_size;
    block_size.x = BLOCK_SIZE_X;
    block_size.y = BLOCK_SIZE_Y;
    grid_size.x = W_x_H / (block_size.x * block_size.y) + (W_x_H % (block_size.x * block_size.y) == 0? 0 : 1);

    // reserva memoria en el device
    hipMalloc((void**)&dev_img, img_size);
    hipMemset(dev_img, 0, img_size);

    // copia los parametros al device
    hipMemcpyToSymbol(HIP_SYMBOL(dev_iter), &iter, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_x_min), &x_min, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_y_min), &y_min, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_x_step), &x_step, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_y_step), &y_step, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_W_x_H), &W_x_H, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_width), &width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_row_size), &row_size, sizeof(int));

    // ejecuta el kernel
    mandelbrot<<<grid_size, block_size>>>(dev_img);
    // copia el resultado al host
    hipMemcpy(hst_img, dev_img, img_size, hipMemcpyDeviceToHost);
    // libera memoria del device
    hipFree(dev_img);
}

// wrapper para llamar al kernel desde C
extern "C" void kernel_wrapper(char *hst_img, int img_size, int iter, double x_min, double y_min, double x_step, double y_step, int W_x_H, int width, int row_size){
    kernel_mandelbrot(hst_img, img_size, iter, x_min, y_min, x_step, y_step, W_x_H, width, row_size);
}